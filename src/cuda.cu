#include "hip/hip_runtime.h"
/*
 * GraphHPC-2017 Contest
 * Betweenness Centrality Problem
 */

#include <iostream>  // std::cout

#include <hip/hip_runtime.h>
#include <thrust/fill.h>

#include "main.h"

#ifndef THREADS_PER_ITERATION
#define THREADS_PER_ITERATION 64
#endif

#ifndef BLOCKS_PER_ITERATION
#define BLOCKS_PER_ITERATION 2
#endif

#define THREADS_PER_BLOCK THREADS_PER_ITERATION / BLOCKS_PER_ITERATION

using namespace std;


__global__ void kernel(vertex_size_t* d_vertices,
                       edge_size_t*   d_edges,
                       vertex_size_t* d_ends,
                       edge_size_t*   d_indices, 
                       double*        d_result,
                       vertex_size_t* d_order,
                       edge_size_t*   d_p_last,
                       edge_size_t*   d_p_prev,
                       vertex_size_t* d_p_val,
                       edge_size_t*   d_sigma,
                       vertex_size_t* d_depth,
                       bool*          d_visited,
                       double*        d_delta,
                       vertex_size_t* d_queue) {
    int ID = threadIdx.x + blockIdx.x * blockDim.x;

    vertex_size_t vertices = *d_vertices;
    edge_size_t   edges    = *d_edges;

    if (ID >= vertices) {
        return;
    }

    d_result  += vertices * ID;

    d_order   += vertices * ID;
    d_p_last  += vertices * ID;
    d_p_prev  += edges    * ID + 1;
    d_p_val   += edges    * ID + 1;
    d_sigma   += vertices * ID;
    d_depth   += vertices * ID;
    d_visited += vertices * ID;
    d_delta   += vertices * ID;
    d_queue   += vertices * ID;

    for (vertex_size_t s = ID; s < vertices; s += THREADS_PER_ITERATION) {
        memset(d_p_last, 0, sizeof(edge_size_t) * vertices);
        memset(d_sigma,  0, sizeof(edge_size_t) * vertices);

        thrust::fill(d_visited, d_visited + vertices, false);
        thrust::fill(d_delta,   d_delta   + vertices, 0.0);

        vertex_size_t order_pos   = 0;
        vertex_size_t p_pos       = 0;
        vertex_size_t queue_front = 0;
        vertex_size_t queue_back  = 0;

        d_sigma[s]            = 1;
        d_depth[s]            = 0;
        d_visited[s]          = true;
        d_queue[queue_back++] = s;

        while (queue_front != queue_back) {
            vertex_size_t v = d_queue[queue_front++];
            d_order[order_pos++] = v;

            for (edge_size_t i = d_indices[v]; i < d_indices[v + 1]; i++) {
                vertex_size_t t = d_ends[i];

                if (!d_visited[t]) {
                    d_depth[t] = d_depth[v] + 1;
                    d_queue[queue_back++] = t;
                    d_visited[t] = true;
                }

                if (d_depth[t] > d_depth[v] && d_depth[t] == d_depth[v] + 1) {
                    d_sigma[t] += d_sigma[v];

                    d_p_prev[p_pos] = d_p_last[t];
                    d_p_val[p_pos] = v;
                    d_p_last[t] = p_pos;
                    p_pos++;
                }
            }
        }

        while (order_pos --> 1)  {
            vertex_size_t v = d_order[order_pos];
            edge_size_t i = d_p_last[v];
            double d = (1 + d_delta[v]) / (double) d_sigma[v];

            while (i != 0) {
                vertex_size_t t = d_p_val[i];
                d_delta[t] += d_sigma[t] * d;
                i = d_p_prev[i];
            }

            d_result[v] += d_delta[v] / 2;
        }
    }
}

/**
 * Calculates betweenness centrality by the algorithm with CUDA stack.
 */
void run(vertex_size_t  h_vertices, 
         edge_size_t    h_edges, 
         vertex_size_t* h_ends, 
         edge_size_t*   h_indices, 
         double*        result) {

    if (THREADS_PER_ITERATION % BLOCKS_PER_ITERATION != 0) {
        // you're idiot;
        exit(1);
    }

    // Allocatins for the graph data

    vertex_size_t* d_vertices;
    edge_size_t*   d_edges;
    vertex_size_t* d_ends;
    edge_size_t*   d_indices;

    hipMalloc((void **) &d_vertices, sizeof(vertex_size_t));
    hipMalloc((void **) &d_edges,    sizeof(edge_size_t));
    hipMalloc((void **) &d_ends,     sizeof(vertex_size_t) * h_edges);
    hipMalloc((void **) &d_indices,  sizeof(edge_size_t) * (h_vertices + 1));

    hipMemcpy(d_vertices, &h_vertices, sizeof(vertex_size_t),                  hipMemcpyHostToDevice);
    hipMemcpy(d_edges,    &h_edges,    sizeof(edge_size_t),                    hipMemcpyHostToDevice);
    hipMemcpy(d_ends,      h_ends,     sizeof(vertex_size_t) * h_edges,        hipMemcpyHostToDevice);
    hipMemcpy(d_indices,   h_indices,  sizeof(edge_size_t) * (h_vertices + 1), hipMemcpyHostToDevice);


    // Allocating an array for results

    double* h_result = (double*) malloc(sizeof(double) * 
                                        h_vertices * 
                                        THREADS_PER_ITERATION);
    double* d_result;

    hipMalloc((void **) &d_result, sizeof(double) * 
                                    h_vertices * 
                                    THREADS_PER_ITERATION);
    hipMemset(d_result, 0, sizeof(double) * 
                            h_vertices * 
                            THREADS_PER_ITERATION);


    // Allocating an temporary arrays for calculations

    vertex_size_t* d_order;
    edge_size_t*   d_p_last;
    edge_size_t*   d_p_prev;
    vertex_size_t* d_p_val;
    edge_size_t*   d_sigma;
    vertex_size_t* d_depth;
    bool*          d_visited;
    double*        d_delta;
    vertex_size_t* d_queue;

    hipMalloc((void **) &d_order,   sizeof(vertex_size_t) * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_p_last,  sizeof(edge_size_t)   * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_p_prev,  sizeof(edge_size_t)   * (h_edges   * THREADS_PER_ITERATION + 1));
    hipMalloc((void **) &d_p_val,   sizeof(vertex_size_t) * (h_edges   * THREADS_PER_ITERATION + 1));
    hipMalloc((void **) &d_sigma,   sizeof(edge_size_t)   * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_depth,   sizeof(vertex_size_t) * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_visited, sizeof(bool)          * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_delta,   sizeof(double)        * h_vertices * THREADS_PER_ITERATION);
    hipMalloc((void **) &d_queue,   sizeof(vertex_size_t) * h_vertices * THREADS_PER_ITERATION);

    // Calculating

    kernel<<<BLOCKS_PER_ITERATION, THREADS_PER_BLOCK>>>(d_vertices, 
                                                        d_edges,
                                                        d_ends, 
                                                        d_indices,
                                                        d_result,
                                                        d_order,
                                                        d_p_last,
                                                        d_p_prev,
                                                        d_p_val,
                                                        d_sigma,
                                                        d_depth,
                                                        d_visited,
                                                        d_delta,
                                                        d_queue);
    hipDeviceSynchronize();

    // Result

    hipMemcpy(h_result, 
               d_result, 
               sizeof(double) * h_vertices * THREADS_PER_ITERATION,
               hipMemcpyDeviceToHost);

    for (vertex_size_t i = 0; i < h_vertices * THREADS_PER_ITERATION; i++) {
        result[i % h_vertices] += h_result[i];
    }

    // Cleaning

    hipFree(d_order);
    hipFree(d_p_last);
    hipFree(d_p_prev);
    hipFree(d_p_val);
    hipFree(d_sigma);
    hipFree(d_depth);
    hipFree(d_visited);
    hipFree(d_delta);
    hipFree(d_queue);

    hipFree(d_result);
    hipFree(d_indices);
    hipFree(d_ends);
    hipFree(d_edges);
    hipFree(d_vertices);

    free(h_result);
}
