#include "hip/hip_runtime.h"
/*
 * GraphHPC-2017 Contest
 * Betweenness Centrality Problem
 */

#include <cstring>  // std::memset
#include <iomanip>  // std::setprecision
#include <iostream> // std::cout, std::fixed

#ifdef DEBUG_SOLUTION
#include <map>      // std::map
#endif

#include <hip/hip_runtime.h>
#include <omp.h>

#include "main.h"

#if defined(CLOCK_MONOTONIC)
#define CLOCK CLOCK_MONOTONIC
#elif defined(CLOCK_REALTIME)
#define CLOCK CLOCK_REALTIME
#else
#error "Failed to find a timing clock."
#endif

#ifndef BLOCKS_PER_ITERATION
#define BLOCKS_PER_ITERATION 64
#endif

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 256
#endif

#define BPI BLOCKS_PER_ITERATION
#define TPB THREADS_PER_BLOCK

#define CPU
#define GPU

using namespace std;


#ifdef DEBUG_SOLUTION
static map<string, timespec> timers;
static map<string, double> timings;
#endif

static inline void timer_start(const string &name) {
#ifdef DEBUG_SOLUTION
    clock_gettime(CLOCK, &timers[name]);
#endif
}

static inline void timer_end(const string &name) {
#ifdef DEBUG_SOLUTION
    timespec finish_time;
    clock_gettime(CLOCK, &finish_time);

    timespec start_time = timers[name];

    double time = finish_time.tv_sec - (double) start_time.tv_sec + 
        (finish_time.tv_nsec - (double) start_time.tv_nsec) * 1.0e-9;
    timings[name] += time;
#endif
}

#ifdef GPU
static __device__ __inline__ double atomic_add_double(double* address, 
                                                      double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address; 
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull,
                        assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}
#endif

#ifdef GPU
__global__ void kernel(vertex_size_t* d_vertices,
                       edge_size_t*   d_edges,
                       vertex_size_t* d_ends,
                       edge_size_t*   d_indices,
                       vertex_size_t* d_order,
                       edge_size_t*   d_p_last,
                       edge_size_t*   d_p_prev,
                       vertex_size_t* d_p_val,
                       edge_size_t*   d_sigma,
                       vertex_size_t* d_depth,
                       int*           d_visited,
                       double*        d_delta,
                       vertex_size_t* d_queue,
                       double*        d_result,
                       double*        d_result_block,
                       vertex_size_t  gpu_start,
                       vertex_size_t  gpu_end) {
    // Offsetting
    
    vertex_size_t  vertices     = *d_vertices;
    edge_size_t    edges        = *d_edges;

    vertex_size_t* order        = d_order        + vertices * blockIdx.x * 2; // magic
    edge_size_t*   p_last       = d_p_last       + vertices * blockIdx.x;
    edge_size_t*   p_prev       = d_p_prev       + edges    * blockIdx.x + 1;
    vertex_size_t* p_val        = d_p_val        + edges    * blockIdx.x + 1;
    edge_size_t*   sigma        = d_sigma        + vertices * blockIdx.x;
    vertex_size_t* depth        = d_depth        + vertices * blockIdx.x;
    int*           visited      = d_visited      + vertices * blockIdx.x;
    double*        delta        = d_delta        + vertices * blockIdx.x;
    vertex_size_t* queue        = d_queue        + vertices * blockIdx.x;

    double*        result_block = d_result_block + vertices * blockIdx.x;

    __syncthreads();

    // Parallel cycle

    for (vertex_size_t s = gpu_start + blockIdx.x; 
         s < gpu_end;
         s += gridDim.x) {
        __shared__ vertex_size_t* order_pos;
        __shared__ edge_size_t    p_pos;
        __shared__ vertex_size_t* queue_front;
        __shared__ vertex_size_t* queue_back;
        __shared__ vertex_size_t  queue_back_cumulative;
        __shared__ vertex_size_t  level; // depth, as a constant

        __shared__ vertex_size_t  need;

        __syncthreads();

        for (vertex_size_t i = threadIdx.x; i < vertices; i += blockDim.x) {
            p_last[i]  = 0;
            sigma[i]   = 0;
            depth[i]   = vertices;
            visited[i] = 0;
            delta[i]   = 0;
        }

        __syncthreads();

        if (threadIdx.x == 0) {
            order_pos     = order;
            p_pos         = 0;
            queue_front   = queue;
            queue_back    = queue;
            level         = 0;

            sigma[s]      = 1;
            depth[s]      = 0;
            visited[s]    = 1;
            *queue_back++ = s;
        }

        __syncthreads();

        while (queue_front != queue_back) {
            if (threadIdx.x == 0) {
                queue_back_cumulative = 0;

                need                  = queue_back - queue_front;
            }

            __syncthreads();

            for (vertex_size_t i = threadIdx.x; i < need; i += blockDim.x) {
                vertex_size_t v  = *(queue_front + i);
                *(order_pos + i) = v;

                for (edge_size_t j = d_indices[v]; j < d_indices[v + 1]; j++) {
                    vertex_size_t t = d_ends[j];

                    if (0 == atomicCAS(visited + t, 0, 1)) {
                        atomicExch(depth + t, level + 1);
                        *(queue_back + atomicAdd(&queue_back_cumulative, 
                                                 (vertex_size_t) 1)) = t;
                    }

                    if (depth[t] > level) {
                        atomicAdd(sigma + t, sigma[v]);
                        edge_size_t p_pos_cache = atomicAdd(&p_pos, 1);
                        p_prev[p_pos_cache] = atomicExch(p_last + t, 
                                                         p_pos_cache);
                        p_val[p_pos_cache] = v;
                    }
                }
            }

            __syncthreads();

            if (threadIdx.x == 0) {
                order_pos    += need;
                queue_front  += need;
                queue_back   += queue_back_cumulative;
                level        ++;
                *order_pos++  = need;
            }

            __threadfence_block();
            __syncthreads();
        }

        __syncthreads();
        
        if (threadIdx.x == 0) {
            // Set to zero the first level of depth (where is only `s` vertex)
            order[1] = 0;

            // Set `order_pos` to a valid cell for the reverse iteration
            order_pos--;
        }

        __syncthreads();

        while (true) {
            if (threadIdx.x == 0) {
                need = *order_pos--;
            }

            __threadfence_block();
            __syncthreads();

            if (need == 0) {
                break;
            }

            for (vertex_size_t i = threadIdx.x; i < need; i += blockDim.x) {
                vertex_size_t v = *(order_pos - i);
                edge_size_t j = p_last[v];
                double d = (1 + delta[v]) / (double) sigma[v];

                while (j != 0) {
                    vertex_size_t t = p_val[j];
                    atomic_add_double(delta + t, sigma[t] * d);
                    j = p_prev[j];
                }

                result_block[v] += delta[v];
            }

            __syncthreads();

            if (threadIdx.x == 0) {
                order_pos -= need;
            }

            __syncthreads();
        }

        __threadfence_block();
        __syncthreads();
    }

    __threadfence_block();
    __syncthreads();

    // Result reduction

    for (vertex_size_t v = threadIdx.x; v < vertices; v += blockDim.x) {
        atomic_add_double(d_result + v, result_block[v]);
    }

    __threadfence_block();
}
#endif

/**
 * Calculates betweenness centrality by the algorithm with mixed
 * OpenMP-CUDA stack.
 */
void run(vertex_size_t  vertices,
         edge_size_t    edges,
         vertex_size_t* ends,     // don't touch the ends array!
         edge_size_t*   indices,  // don't touch the indices array!
         double*        result) { // be careful!

    //
    // Pre-init
    //

    #ifdef GPU
    timer_start("GPU Pre-initialization");
    hipSetDevice(0);
    timer_end("GPU Pre-initialization");
    #endif


    //
    // Constants
    //

    // CPU

    #ifdef CPU

    unsigned threads = omp_get_max_threads();

    vertex_size_t* order;
    edge_size_t*   p_last;
    edge_size_t*   p_prev;
    vertex_size_t* p_val;
    edge_size_t*   sigma;
    vertex_size_t* depth;
    double*        delta;
    vertex_size_t* queue;

    double*        result_thread;

    #endif

    #ifdef GPU
    double*        h_result;
    #endif

    // GPU

    #ifdef GPU

    vertex_size_t* d_vertices;
    edge_size_t*   d_edges;
    vertex_size_t* d_ends;
    edge_size_t*   d_indices;

    vertex_size_t* d_order;
    edge_size_t*   d_p_last;
    edge_size_t*   d_p_prev;
    vertex_size_t* d_p_val;
    edge_size_t*   d_sigma;
    vertex_size_t* d_depth;
    int*           d_visited;
    double*        d_delta;
    vertex_size_t* d_queue;

    double*        d_result;
    double*        d_result_block;

    #endif


    //
    // Allocations
    //

    timer_start("Allocations");

    // # linter-disable: length-limit
    #pragma omp parallel sections
    {
        // CPU
        #pragma omp section
        {
            #ifdef CPU
            order         = new vertex_size_t[threads * vertices];
            p_last        = new   edge_size_t[threads * vertices];
            p_prev        = new   edge_size_t[threads * (edges + 1)];
            p_val         = new vertex_size_t[threads * (edges + 1)];
            sigma         = new   edge_size_t[threads * vertices];
            depth         = new vertex_size_t[threads * vertices];
            delta         = new        double[threads * vertices];
            queue         = new vertex_size_t[threads * vertices];

            result_thread = new        double[threads * vertices];
            #endif

            #ifdef GPU
            h_result      = new        double[vertices];
            #endif

            #ifdef CPU
            memset(result_thread, 0, sizeof(double) * threads * vertices);
            #endif
        }

        // GPU
        #ifdef GPU
        #pragma omp section
        {
            hipMalloc((void **) &d_vertices, sizeof(vertex_size_t));
            hipMalloc((void **) &d_edges,    sizeof(edge_size_t));
            hipMalloc((void **) &d_ends,     sizeof(vertex_size_t) * edges);
            hipMalloc((void **) &d_indices,  sizeof(edge_size_t)   * (vertices + 1));

            // Array with a level-ordering.
            // Each level contains some vertex indices, and number of indices,
            // as a last element in a level.
            // This structure allows us to iterate for each level in reverse 
            // ordering.
            hipMalloc((void **) &d_order,   sizeof(vertex_size_t) * vertices * BPI * 2);

            hipMalloc((void **) &d_p_last,  sizeof(edge_size_t)   * vertices * BPI);
            hipMalloc((void **) &d_p_prev,  sizeof(edge_size_t)   * (edges   * BPI + 1));
            hipMalloc((void **) &d_p_val,   sizeof(vertex_size_t) * (edges   * BPI + 1));
            hipMalloc((void **) &d_sigma,   sizeof(edge_size_t)   * vertices * BPI);
            hipMalloc((void **) &d_depth,   sizeof(vertex_size_t) * vertices * BPI);
            hipMalloc((void **) &d_visited, sizeof(int)           * vertices * BPI);
            hipMalloc((void **) &d_delta,   sizeof(double)        * vertices * BPI);
            hipMalloc((void **) &d_queue,   sizeof(vertex_size_t) * vertices * BPI);

            hipMalloc((void **) &d_result,       sizeof(double) * vertices);
            hipMalloc((void **) &d_result_block, sizeof(double) * vertices * BPI);

            hipMemcpy(d_vertices, &vertices, sizeof(vertex_size_t),                  hipMemcpyHostToDevice);
            hipMemcpy(d_edges,    &edges,    sizeof(edge_size_t),                    hipMemcpyHostToDevice);
            hipMemcpy(d_ends,      ends,     sizeof(vertex_size_t) * edges,          hipMemcpyHostToDevice);
            hipMemcpy(d_indices,   indices,  sizeof(edge_size_t)   * (vertices + 1), hipMemcpyHostToDevice);

            hipMemset(d_result,       0, sizeof(double) * vertices);
            hipMemset(d_result_block, 0, sizeof(double) * vertices * BPI);
        }
        #endif
    }

    timer_end("Allocations");


    //
    // Partitioning
    //

    // Splits for CPU 2/3 and GPU 1/3 of all vertices

    #if defined(CPU) && defined(GPU)
    vertex_size_t cpu_start = 0;
    vertex_size_t cpu_end   = vertices / 3 * 2;
    vertex_size_t gpu_start = vertices / 3 * 2;
    vertex_size_t gpu_end   = vertices;
    #elif defined(CPU)
    vertex_size_t cpu_start = 0
    vertex_size_t cpu_end   = vertices;
    #elif defined(GPU)
    vertex_size_t gpu_start = 0;
    vertex_size_t gpu_end   = vertices;
    #else
    #error "What a strange code, that does nothing? :("
    #endif


    //
    // Main logic
    //

    #ifdef GPU
    timer_start("GPU main logic");
    kernel<<<BLOCKS_PER_ITERATION, THREADS_PER_BLOCK>>>(d_vertices, 
                                                        d_edges,
                                                        d_ends, 
                                                        d_indices,
                                                        d_order,
                                                        d_p_last,
                                                        d_p_prev,
                                                        d_p_val,
                                                        d_sigma,
                                                        d_depth,
                                                        d_visited,
                                                        d_delta,
                                                        d_queue,
                                                        d_result,
                                                        d_result_block,
                                                        gpu_start,
                                                        gpu_end);
    #endif

    // GPU is working async, that means, we can calculate some data on CPU

    #ifdef CPU
    timer_start("CPU main logic");

    #pragma omp parallel firstprivate(order, p_last, p_prev, p_val, sigma, depth, delta, queue, result_thread)
    {
        // Constants

        unsigned tid = omp_get_thread_num();

        // Offsetting

        order         += vertices * tid;
        p_last        += vertices * tid;
        p_prev        += edges    * tid + 1;
        p_val         += edges    * tid + 1;
        sigma         += vertices * tid;
        depth         += vertices * tid;
        delta         += vertices * tid;
        queue         += vertices * tid;

        result_thread += vertices * tid;

        // Variables

        vertex_size_t* order_pos;
        edge_size_t    p_pos;
        vertex_size_t* queue_front;
        vertex_size_t* queue_back;

        // Parallel cycle

        #pragma omp for
        for (vertex_size_t s = cpu_start; s < cpu_end; ++s) {
            memset(p_last, 0, sizeof(edge_size_t)   * vertices);
            memset(sigma,  0, sizeof(edge_size_t)   * vertices);
            memset(depth,  0, sizeof(vertex_size_t) * vertices);
            memset(delta,  0, sizeof(double)        * vertices);

            order_pos    = order;
            p_pos        = 1;
            queue_front  = queue;
            queue_back   = queue;

            sigma[s]      = 1;
            depth[s]      = 1;
            *queue_back++ = s;

            while (queue_front != queue_back) {
                vertex_size_t v = *queue_front++;
                *order_pos++ = v;

                for (vertex_size_t *t = &ends[indices[v]], 
                                   *r = &ends[indices[v + 1]];
                     t != r; 
                     ++t) {
                    if (depth[*t] == 0) {
                        depth[*t] = depth[v] + 1;
                        *queue_back++ = *t;
                    }

                    if (depth[*t] > depth[v]) {
                        sigma[*t] += sigma[v];

                        p_prev[p_pos] = p_last[*t];
                        p_val[p_pos] = v;
                        p_last[*t] = p_pos++;
                    }
                }
            }

            while (--order_pos != order)  {
                edge_size_t i = p_last[*order_pos];
                double d = (1 + delta[*order_pos]) / (double) sigma[*order_pos];

                while (i != 0) {
                    delta[p_val[i]] += sigma[p_val[i]] * d;
                    i = p_prev[i];
                }

                result_thread[*order_pos] += delta[*order_pos];
            }
        }

        // Result reduction

        for (vertex_size_t v = 0; v < vertices; v++) {
            #pragma omp atomic
            result[v] += result_thread[v];
        }
    }

    timer_end("CPU main logic");
    #endif

    // Waiting for GPU
    #ifdef GPU
    hipDeviceSynchronize();
    timer_end("GPU main logic");
    #endif


    //
    // Result calculation
    //

    #ifdef GPU
    timer_start("GPU result reduction");

    hipMemcpy(h_result, 
               d_result, 
               sizeof(double) * vertices,
               hipMemcpyDeviceToHost);

    #pragma omp parallel for
    for (vertex_size_t v = 0; v < vertices; v++) {
        result[v] += h_result[v];
    }

    timer_end("GPU result reduction");
    #endif

    #pragma omp parallel for
    for (vertex_size_t v = 0; v < vertices; v++) {
        result[v] /= 2;
    }


    //
    // Cleaning
    // Deal with it!
    //

    timer_start("Cleaning");

    #pragma omp parallel sections
    {
        // CPU
        #pragma omp section
        {
            #ifdef GPU
            delete[] h_result;
            #endif

            #ifdef CPU
            delete[] result_thread;

            delete[] queue;
            delete[] delta;
            delete[] depth;
            delete[] sigma;
            delete[] p_val;
            delete[] p_prev;
            delete[] p_last;
            delete[] order;
            #endif
        }

        // GPU
        #ifdef GPU
        #pragma omp section
        {
            hipDeviceReset();

            /*
            hipFree(d_result_block);
            hipFree(d_result);

            hipFree(d_queue);
            hipFree(d_delta);
            hipFree(d_visited);
            hipFree(d_depth);
            hipFree(d_sigma);
            hipFree(d_p_val);
            hipFree(d_p_prev);
            hipFree(d_p_last);
            hipFree(d_order);

            hipFree(d_indices);
            hipFree(d_ends);
            hipFree(d_edges);
            hipFree(d_vertices);
            */
        }
        #endif
    }

    timer_end("Cleaning");


    //
    // Debug / Timings
    //

    #ifdef DEBUG_SOLUTION

    cout << endl;

    for (auto &it : timings) {
        cout << " - " << it.first << ": ";
        cout << setprecision(5) << fixed << it.second << " sec.";
        cout << endl;
    }

    timers.clear();
    timings.clear();

    #endif
}
