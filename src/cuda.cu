#include "hip/hip_runtime.h"
/*
 * GraphHPC-2017 Contest
 * Betweenness Centrality Problem
 */

#include <stdio.h> // std::printf

#include <hip/hip_runtime.h>

#include "main.h"

#ifndef BLOCKS_PER_ITERATION
#define BLOCKS_PER_ITERATION 64
#endif

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 512
#endif

using namespace std;


__device__ __inline__ double atomic_add_double(double* address, 
                                               double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address; 
    unsigned long long int old = *address_as_ull, assumed;
    
    do {
        assumed = old;
        old = atomicCAS(address_as_ull,
                        assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
    
    return __longlong_as_double(old);
}

__global__ void kernel_prepare(vertex_size_t* d_vertices,
                               double*        d_result,
                               double*        d_result_block) {
    vertex_size_t vertices = *d_vertices;

    double* result_block = d_result_block + vertices * blockIdx.x;

    for (vertex_size_t i = threadIdx.x; i < vertices; i += blockDim.x) {
        result_block[i] = 0;
    }

    if (blockIdx.x == 0) {
        for (vertex_size_t i = threadIdx.x; i < vertices; i += blockDim.x) {
            d_result[i] = 0;
        }
    }

    __threadfence_block();
}

__global__ void kernel_calculate_threads_optimum(vertex_size_t* d_vertices,
                                                 edge_size_t*   d_edges,
                                                 vertex_size_t* d_ends,
                                                 edge_size_t*   d_indices) {
    // nothing...
}

__global__ void kernel_summarize(vertex_size_t* d_vertices,
                                 double*        d_result,
                                 double*        d_result_block) {
    vertex_size_t vertices = *d_vertices;

    double* result_block = d_result_block + vertices * blockIdx.x;

    for (vertex_size_t v = threadIdx.x; v < vertices; v += blockDim.x) {
        atomic_add_double(d_result + v, result_block[v]);
    }

    __threadfence_block();
}

__global__ void kernel_debug_graph_data(vertex_size_t* d_vertices,
                                        edge_size_t*   d_edges,
                                        vertex_size_t* d_ends,
                                        edge_size_t*   d_indices) {
    vertex_size_t vertices = *d_vertices;
    edge_size_t   edges    = *d_edges;

    printf("\n\nGPU GRAPH DATA\n");

    printf("Vertices: %d\n", vertices);
    printf("Edges:    %lld\n", edges);

    printf("Ends:     ");
    for (edge_size_t i = 0; i < edges; i++) {
        printf("%d ", d_ends[i]);
    }

    printf("\n");

    printf("Indices:   ");
    for (vertex_size_t i = 0; i < vertices + 1; i++) {
        printf("%lld ", d_indices[i]);
    }

    printf("\n\n");
}

__global__ void kernel_debug_result_data(vertex_size_t* d_vertices,
                                         double*        d_result,
                                         double*        d_result_block) {
    vertex_size_t vertices = *d_vertices;

    printf("\n\nGPU RESULT DATA\n");

    printf("Global:   ");
    for (vertex_size_t v = 0; v < vertices; v++) {
        printf("%f ", d_result[v]);
    }

    for (unsigned block = 0; block < BLOCKS_PER_ITERATION; block++) {
        printf("Block #%d: ", block);

        double* result_block = d_result_block + vertices * block;
        for (vertex_size_t v = 0; v < vertices; v++) {
            printf("%f ", result_block[v]);
        }

        printf("\n");
    }

    printf("\n\n");
}

__global__ void kernel_calculate_bc(vertex_size_t* d_vertices,
                                    edge_size_t*   d_edges,
                                    vertex_size_t* d_ends,
                                    edge_size_t*   d_indices,
                                    double*        d_result_block,
                                    vertex_size_t* d_order,
                                    edge_size_t*   d_p_last,
                                    edge_size_t*   d_p_prev,
                                    vertex_size_t* d_p_val,
                                    edge_size_t*   d_sigma,
                                    vertex_size_t* d_depth,
                                    int*           d_visited,
                                    double*        d_delta,
                                    vertex_size_t* d_queue) {
    vertex_size_t  vertices     = *d_vertices;
    edge_size_t    edges        = *d_edges;

    double*        result_block = d_result_block + vertices * blockIdx.x;

    vertex_size_t* order        = d_order        + vertices * blockIdx.x * 2; // magic
    edge_size_t*   p_last       = d_p_last       + vertices * blockIdx.x;
    edge_size_t*   p_prev       = d_p_prev       + edges    * blockIdx.x + 1;
    vertex_size_t* p_val        = d_p_val        + edges    * blockIdx.x + 1;
    edge_size_t*   sigma        = d_sigma        + vertices * blockIdx.x;
    vertex_size_t* depth        = d_depth        + vertices * blockIdx.x;
    int*           visited      = d_visited      + vertices * blockIdx.x;
    double*        delta        = d_delta        + vertices * blockIdx.x;
    vertex_size_t* queue        = d_queue        + vertices * blockIdx.x;

    __syncthreads();

    for (vertex_size_t s = blockIdx.x; s < vertices; s += gridDim.x) {
        __shared__ vertex_size_t* order_pos;
        __shared__ edge_size_t    p_pos;
        __shared__ vertex_size_t* queue_front;
        __shared__ vertex_size_t* queue_back;
        __shared__ vertex_size_t  queue_back_cumulative;
        __shared__ vertex_size_t  level; // depth, as a constant

        __shared__ vertex_size_t  need;

        __syncthreads();

        for (vertex_size_t i = threadIdx.x; i < vertices; i += blockDim.x) {
            p_last[i]  = 0;
            sigma[i]   = 0;
            depth[i]   = vertices;
            visited[i] = 0;
            delta[i]   = 0;
        }

        __syncthreads();

        if (threadIdx.x == 0) {
            order_pos     = order;
            p_pos         = 0;
            queue_front   = queue;
            queue_back    = queue;
            level         = 0;

            sigma[s]      = 1;
            depth[s]      = 0;
            visited[s]    = 1;
            *queue_back++ = s;
        }

        __syncthreads();

        while (queue_front != queue_back) {
            if (threadIdx.x == 0) {
                queue_back_cumulative = 0;

                need                  = queue_back - queue_front;
            }

            __syncthreads();

            for (vertex_size_t i = threadIdx.x; i < need; i += blockDim.x) {
                vertex_size_t v  = *(queue_front + i);
                *(order_pos + i) = v;

                for (edge_size_t j = d_indices[v]; j < d_indices[v + 1]; j++) {
                    vertex_size_t t = d_ends[j];

                    if (0 == atomicCAS(visited + t, 0, 1)) {
                        atomicExch(depth + t, level + 1);
                        *(queue_back + atomicAdd(&queue_back_cumulative, 
                                                 (vertex_size_t) 1)) = t;
                    }

                    if (depth[t] > level) {
                        atomicAdd(sigma + t, sigma[v]);
                        edge_size_t p_pos_cache = atomicAdd(&p_pos, 1);
                        p_prev[p_pos_cache] = atomicExch(p_last + t, 
                                                         p_pos_cache);
                        p_val[p_pos_cache] = v;
                    }
                }
            }

            __syncthreads();

            if (threadIdx.x == 0) {
                order_pos    += need;
                queue_front  += need;
                queue_back   += queue_back_cumulative;
                level        ++;
                *order_pos++  = need;
            }

            __threadfence_block();
            __syncthreads();
        }

        __syncthreads();
        
        if (threadIdx.x == 0) {
            // Set to zero the first level of depth (where is only `s` vertex)
            order[1] = 0;

            // Set `order_pos` to a valid cell for the reverse iteration
            order_pos--;
        }

        #ifdef DEBUG_SOLUTION
        if (blockIdx.x == 0 && threadIdx.x == 0) {
            printf("\n\n");

            printf("P_LAST:\n");
            for (vertex_size_t i = 0; i < vertices; i++) {
                printf("%d: %lld | ", i, p_last[i]);
            }

            printf("\n\n");

            printf("P_PREV:\n");
            for (edge_size_t i = 0; i < p_pos; i++) {
                printf("%d: %lld | ", i, p_prev[i]);
            }

            printf("\n\n");

            printf("P_VAL:\n");
            for (edge_size_t i = 0; i < p_pos; i++) {
                printf("%d: %lld | ", i, p_val[i]);
            }

            printf("\n\n");

            printf("SIGMA:\n");
            for (edge_size_t i = 0; i < vertices; i++) {
                printf("%d: %lld | ", i, sigma[i]);
            }

            printf("\n\n");

            printf("DEPTH:\n");
            for (edge_size_t i = 0; i < vertices; i++) {
                printf("%d: %lld | ", i, depth[i]);
            }

            printf("\n\n");
        }
        #endif // DEBUG_SOLUTION

        __syncthreads();

        while (true) {
            if (threadIdx.x == 0) {
                need = *order_pos--;
            }

            __threadfence_block();
            __syncthreads();

            if (need == 0) {
                break;
            }

            for (vertex_size_t i = threadIdx.x; i < need; i += blockDim.x) {
                vertex_size_t v = *(order_pos - i);
                edge_size_t j = p_last[v];
                double d = (1 + delta[v]) / (double) sigma[v];

                while (j != 0) {
                    vertex_size_t t = p_val[j];
                    atomic_add_double(delta + t, sigma[t] * d);
                    j = p_prev[j];
                }

                result_block[v] += delta[v] / 2;
            }

            __syncthreads();

            if (threadIdx.x == 0) {
                order_pos -= need;
            }

            __syncthreads();
        }

        __threadfence_block();
        __syncthreads();
    }
}

/**
 * Calculates betweenness centrality by the algorithm with CUDA stack.
 */
void run(vertex_size_t  h_vertices, 
         edge_size_t    h_edges, 
         vertex_size_t* ends, 
         edge_size_t*   indices, 
         double*        h_result) {

    // Allocations for the graph data

    vertex_size_t* d_vertices;
    edge_size_t*   d_edges;
    vertex_size_t* d_ends;
    edge_size_t*   d_indices;

    hipMalloc((void **) &d_vertices, sizeof(vertex_size_t));
    hipMalloc((void **) &d_edges,    sizeof(edge_size_t));
    hipMalloc((void **) &d_ends,     sizeof(vertex_size_t) * h_edges);
    hipMalloc((void **) &d_indices,  sizeof(edge_size_t) * (h_vertices + 1));

    hipMemcpy(d_vertices, &h_vertices, sizeof(vertex_size_t),                  hipMemcpyHostToDevice);
    hipMemcpy(d_edges,    &h_edges,    sizeof(edge_size_t),                    hipMemcpyHostToDevice);
    hipMemcpy(d_ends,      ends,       sizeof(vertex_size_t) * h_edges,        hipMemcpyHostToDevice);
    hipMemcpy(d_indices,   indices,    sizeof(edge_size_t) * (h_vertices + 1), hipMemcpyHostToDevice);

    #ifdef DEBUG_SOLUTION
    kernel_debug_graph_data<<<1, 1>>>(d_vertices, 
                                      d_edges, 
                                      d_ends, 
                                      d_indices);
    hipDeviceSynchronize();
    #endif // DEBUG_SOLUTION


    // Allocating an arrays for results

    double* d_result;
    double* d_result_block;

    hipMalloc((void **) &d_result, sizeof(double) * h_vertices);
    hipMalloc((void **) &d_result_block, sizeof(double) * 
                                          h_vertices * 
                                          BLOCKS_PER_ITERATION);

    #ifndef DEBUG_SOLUTION
    hipMemset(d_result, 0, sizeof(double) * h_vertices);
    hipMemset(d_result_block, 0, sizeof(double) * 
                                h_vertices *
                                BLOCKS_PER_ITERATION);
    #endif // DEBUG_SOLUTION


    // Allocating an temporary arrays for main calculations

    vertex_size_t* d_order;
    edge_size_t*   d_p_last;
    edge_size_t*   d_p_prev;
    vertex_size_t* d_p_val;
    edge_size_t*   d_sigma;
    vertex_size_t* d_depth;
    int*           d_visited;
    double*        d_delta;
    vertex_size_t* d_queue;

    // Array with a level-ordering.
    // Each level contains some vertex indices, and number of indices,
    // as a last element in a level.
    // This structure allows us to iterate for each level in reverse 
    // ordering.
    hipMalloc((void **) &d_order,   sizeof(vertex_size_t) * h_vertices * BLOCKS_PER_ITERATION * 2);

    hipMalloc((void **) &d_p_last,  sizeof(edge_size_t)   * h_vertices * BLOCKS_PER_ITERATION);
    hipMalloc((void **) &d_p_prev,  sizeof(edge_size_t)   * (h_edges   * BLOCKS_PER_ITERATION + 1));
    hipMalloc((void **) &d_p_val,   sizeof(vertex_size_t) * (h_edges   * BLOCKS_PER_ITERATION + 1));
    hipMalloc((void **) &d_sigma,   sizeof(edge_size_t)   * h_vertices * BLOCKS_PER_ITERATION);
    hipMalloc((void **) &d_depth,   sizeof(vertex_size_t) * h_vertices * BLOCKS_PER_ITERATION);
    hipMalloc((void **) &d_visited, sizeof(int)           * h_vertices * BLOCKS_PER_ITERATION);
    hipMalloc((void **) &d_delta,   sizeof(double)        * h_vertices * BLOCKS_PER_ITERATION);
    hipMalloc((void **) &d_queue,   sizeof(vertex_size_t) * h_vertices * BLOCKS_PER_ITERATION);


    // Calculating

    #ifdef DEBUG_SOLUTION
    kernel_prepare<<<BLOCKS_PER_ITERATION, THREADS_PER_BLOCK>>>(d_vertices,
                                                                d_result, 
                                                                d_result_block);
    hipDeviceSynchronize();
    #endif // DEBUG_SOLUTION

    kernel_calculate_bc<<<BLOCKS_PER_ITERATION, THREADS_PER_BLOCK>>>(d_vertices, 
                                                                     d_edges,
                                                                     d_ends, 
                                                                     d_indices,
                                                                     d_result_block,
                                                                     d_order,
                                                                     d_p_last,
                                                                     d_p_prev,
                                                                     d_p_val,
                                                                     d_sigma,
                                                                     d_depth,
                                                                     d_visited,
                                                                     d_delta,
                                                                     d_queue);
    hipDeviceSynchronize();

    kernel_summarize<<<BLOCKS_PER_ITERATION, THREADS_PER_BLOCK>>>(d_vertices,
                                                                  d_result, 
                                                                  d_result_block);
    hipDeviceSynchronize();

    // Result

    #ifdef DEBUG_SOLUTION
    kernel_debug_result_data<<<1, 1>>>(d_vertices,
                                       d_result, 
                                       d_result_block);
    hipDeviceSynchronize();
    #endif // DEBUG_SOLUTION

    hipMemcpy(h_result, 
               d_result, 
               sizeof(double) * h_vertices,
               hipMemcpyDeviceToHost);

    // Cleaning

    hipFree(d_queue);
    hipFree(d_delta);
    hipFree(d_visited);
    hipFree(d_depth);
    hipFree(d_sigma);
    hipFree(d_p_val);
    hipFree(d_p_prev);
    hipFree(d_p_last);
    hipFree(d_order);

    hipFree(d_result_block);
    hipFree(d_result);

    hipFree(d_indices);
    hipFree(d_ends);
    hipFree(d_edges);
    hipFree(d_vertices);
}
